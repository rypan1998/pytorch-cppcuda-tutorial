#include "hip/hip_runtime.h"
#include <torch/extension.h>


template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    // 如果超过输出范围，则跳过不能算
    if (n>=feats.size(0) || f>=feats.size(2)) return;

    // point -1~1. 需要正规化到 [0,1] 之间
    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;
    
    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;
    // 保存计算结果
    feat_interp[n][f] = (1-u)*(a*feats[n][0][f] +
                               b*feats[n][1][f] +
                               c*feats[n][2][f] +
                               d*feats[n][3][f]) + 
                            u*(a*feats[n][4][f] +
                               b*feats[n][5][f] +
                               c*feats[n][6][f] +
                               d*feats[n][7][f]);
}


torch::Tensor trilinear_fw_cu(
    const torch::Tensor feats,
    const torch::Tensor points
){
    const int N = feats.size(0), F = feats.size(2);
    
    // 定义 feat_interp 作为输出，数据格式、GPU等属性必须要和 feats 保持一致，因此这里直接用 feats.options() 作为所有数据属性的输入即可
    torch::Tensor feat_interp = torch::empty({N, F}, feats.options());
    // 知识拓展：如果没法用 options 而是需要自定义写法（比如最后的输出为整型）？
    // torch::empty({N,F}, torch::dtype(torch::kInt32).device(feats.device));
    // 解释：kInt32 为整型，直接紧跟它之后通过 device 指定 gpu 即可

    const dim3 threads(16, 16);// 这里N和F是可并行的，共两个次元（256不会出错）
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y-1)/threads.y);

    // 建议调用该Kernel的第二个参数为当前函数名，方便报错debug
    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu", 
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return feat_interp;
}


template <typename scalar_t>
__global__ void trilinear_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> dL_dfeat_interp,
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> dL_dfeats
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n>=feats.size(0) || f>=feats.size(2)) return;

    // point -1~1
    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;
    
    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;

    // 八个格点，每个格点的特征值都需要更新
    dL_dfeats[n][0][f] = (1-u)*a*dL_dfeat_interp[n][f];
    dL_dfeats[n][1][f] = (1-u)*b*dL_dfeat_interp[n][f];
    dL_dfeats[n][2][f] = (1-u)*c*dL_dfeat_interp[n][f];
    dL_dfeats[n][3][f] = (1-u)*d*dL_dfeat_interp[n][f];
    dL_dfeats[n][4][f] = u*a*dL_dfeat_interp[n][f];
    dL_dfeats[n][5][f] = u*b*dL_dfeat_interp[n][f];
    dL_dfeats[n][6][f] = u*c*dL_dfeat_interp[n][f];
    dL_dfeats[n][7][f] = u*d*dL_dfeat_interp[n][f];
}

// 把计算得到的 Loss 通过求导的方式，反向传播对 F 进行更新
torch::Tensor trilinear_bw_cu(
    const torch::Tensor dL_dfeat_interp,// Loss对feat_interp的偏导，已知量
    const torch::Tensor feats,
    const torch::Tensor points
){
    const int N = feats.size(0), F = feats.size(2);
    
    // 输出结果：Loss对各顶点特征值的偏导。特征值的维度是 [N,8,F] 那么偏导值也是这个维度
    torch::Tensor dL_dfeats = torch::empty({N, 8, F}, feats.options());

    const dim3 threads(16, 16);
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y-1)/threads.y);

    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_bw_cu", 
    ([&] {
        trilinear_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dfeat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),// 由于 feat_interp 的维度是 [N,F]，所以求偏导后的结果也是这个维度，所以是 2
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            dL_dfeats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dL_dfeats;
}
